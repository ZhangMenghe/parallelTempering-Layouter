#include "hip/hip_runtime.h"
#include <limits.h>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include "layoutConstrains.h"
#include "room.h"
#include "predefinedConstrains.h"
#define RES_NUM 1
using namespace std;
using namespace cv;

const unsigned int nBlocks = 10 ;
const unsigned int WHICH_GPU = 0;
const unsigned int nTimes =20;

void roomInitialization(Room* m_room);

extern __shared__ singleObj sObjs[];
extern __shared__ float gtemparing[];
__device__ __managed__ Room* room;

class automatedLayout
{
private:
	layoutConstrains *constrains;
	float *weights;
	int debugParam = 0;
    void random_along_wall(int furnitureID);
    float cost_function();

public:
    // Room * room;
	float min_cost;
    float *resTransAndRot;
	automatedLayout(vector<float>in_weights);
	void generate_suggestions();
	void display_suggestions();
	__device__ void debugDevice(){
		// printf("id: %d\n", sObjs[0].id);
		// printf("res:\n", resTransAndRot[0]);
		// sObjs[0] = room->deviceObjs[1];
		// debugParam = 2;//room->deviceObjs[1].id;
	}
	__device__ void assignId(){
		printf("%d\n", room->deviceObjs[0].id );
		// sObjs[0] = room->deviceObjs[0];
		// sObjs[0].id = 9;
		// resTransAndRot[0] = 0;
		// printf("%f", resTransAndRot[0]);
	}
};

int seed;
void setUpDevices(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(WHICH_GPU <= deviceCount) {
    hipError_t err = hipSetDevice(WHICH_GPU);
    if(err != hipSuccess)
        cout<< "CUDA error:" <<hipGetErrorString(err)<<endl;
    }
    else {
        cout << "Invalid GPU device " << WHICH_GPU << endl;
        exit(-1);
    }
    int wgpu;
    hipGetDevice(&wgpu);
    hipDeviceReset();
}
void startToProcess(Room * m_room, vector<float> weights){
	//cout<<"hello from mcmc"<<endl;
	setUpDevices();
	roomInitialization(m_room);
	automatedLayout * layout = new automatedLayout(weights);
	layout->generate_suggestions();
   // 	// layout->display_suggestions();
}

__device__ float density_function(float beta, float cost) {
    // printf("%f-%f\n", beta, cost);
	return exp2f(-beta * cost);
}

__device__ float get_randomNum(unsigned int seed, int maxLimit) {
  /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t state;

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);

  /* hiprand works like rand - except that it takes a state as a parameter */
  return hiprand(&state) % maxLimit;
 // int res = hiprand(&state) % maxLimit;
 // printf("%d ", res);
 // return res;
}



__device__ float cost_function_device(float * data, int length){
    //dummy cost, just sum up all
    float res = 0;

    for(int i=0; i<length; i++)
        res += data[i];
    // printf("res: %f\n", res);
    return res/1000;
}
__device__
void changeTemparature(float * temparature, unsigned int seed){
    int t1 = get_randomNum(seed, nBlocks);
    int t2=t1;
    while(t2 == t1)
        t2 = get_randomNum(seed + 100, nBlocks);
    float tmp = temparature[t1];
    temparature[t1] = temparature[t2];
    temparature[t2] = tmp;
}
__device__
void ActualHW(int randTimes, int numofObjs, unsigned int seed, int* pickedIdAddr, float*sArray, float * cost, float *temparature){
    // bool hit = false;
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for(int t=0; t<randTimes; t++){
        if(pickedIdAddr[t] == threadIdx.x){
            if(t % 10 == 0)
                changeTemparature(temparature, seed+index);
            float cost_pri = cost_function_device(sArray, numofObjs);
            float p0 = density_function(temparature[blockIdx.x], cost_pri);
            float tmpKeep = sArray[threadIdx.x];
            sArray[threadIdx.x] = get_randomNum(seed+index, 1000);

            float cost_post = cost_function_device(sArray, numofObjs);
            float p = density_function(temparature[blockIdx.x], cost_post);
            float alpha = min(1.0f, p/p0);
            // printf("p/p0: %f\n", p/p0);
            float t =0.8f;
            //change back
            if(alpha>t)
                sArray[threadIdx.x] = tmpKeep;
            else{
                if(sArray[threadIdx.x]>tmpKeep)
                    printf("%f - %f\n", tmpKeep, sArray[threadIdx.x]);
                cost[blockIdx.x] = cost_post;
            }


            // hit = true;
        }
    }
    // return hit;
}
__global__
void Do_Metropolis_Hastings(unsigned int seed, float * rArray){
	gtemparing[blockIdx.x] = blockIdx.x;//-get_randomNum(seed+blockIdx.x, 100) / 10;
	__syncthreads();
	rArray[blockIdx.x] = gtemparing[blockIdx.x];
}
__global__
void AssignFurnitures(int * pickedIdxs, unsigned int seed){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	sObjs[index] = room->deviceObjs[threadIdx.x];
	__syncthreads();
	printf("%d\n", sObjs[index].id);
}
__global__
void simpleHW(int numofObjs, float * gValues, float* gArray,unsigned int seed,int*pickedIdxs, int randTimes){
    //here should be dynamic shared mem
    //__shared__ float sArray[30];
    extern __shared__ float sharedMem[];
    float * sArray = sharedMem;
    float * lastSumUp = (float *) & sArray[nBlocks*numofObjs];
    float * temparature = (float *) & lastSumUp[nBlocks];
    //initialize
    int startIdx = blockIdx.x * numofObjs;
    int idx =  startIdx+ threadIdx.x;

    sArray[idx] = gValues[threadIdx.x];
    temparature[blockIdx.x] = -get_randomNum(seed+blockIdx.x, 100) / 10;
    // printf("temp: %f", temparature[blockIdx.x]);
    lastSumUp[blockIdx.x] = 0;
    for(int i = 0;i<numofObjs; i++)
        lastSumUp[blockIdx.x] += gValues[i];

    int* pickedIdAddr = &pickedIdxs[blockIdx.x * randTimes];

    ActualHW(randTimes, numofObjs, seed, pickedIdAddr, &sArray[startIdx], lastSumUp, temparature);
    __syncthreads();
    gArray[idx] = sArray[idx];
}
void naiveCUDA(){
	float *gValues;
    float * gArray;
    int * pickedIdxs;

    int numofObjs = 5;

    // int nTimes =20000;

    int totalSize = nBlocks*numofObjs* sizeof(float);

    hipMallocManaged(&gValues, numofObjs * sizeof(float));
    for(int i=0; i<numofObjs; i++)
        gValues[i] = 1000;
    hipMallocManaged(&pickedIdxs, nBlocks*nTimes * sizeof(int));
    for(int i=0; i<nBlocks*nTimes; i++)
        pickedIdxs[i] = rand()%numofObjs;
    // for(int i=0; i<nBlocks*nTimes; i++)
    //     cout<<pickedIdxs[i]<<" ";
    // cout<<endl;

    hipMallocManaged(&gArray, totalSize);
    //dynamic shared mem, <<<nb, nt, sm>>>
    simpleHW<<<nBlocks, numofObjs, totalSize + 2*nBlocks*sizeof(float)>>>(numofObjs, gValues, gArray,time(NULL),pickedIdxs,nTimes);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    for(int i=0;i<nBlocks;i++){
        for(int j=0; j<numofObjs; j++)
            cout<<gArray[i * numofObjs+ j]<<" ";
        cout<<endl;
    }

    // Free memory
    hipFree(gValues);
    hipFree(gArray);
    hipFree(pickedIdxs);
}
void Room::RoomCopy(const Room & m_room){
	objctNum = m_room.objctNum;
	hipMallocManaged(&deviceObjs,  objctNum * sizeof(singleObj));
	for(int i=0; i<objctNum; i++)
		deviceObjs[i] = m_room.objects[i];

}
void roomInitialization(Room* m_room){
	hipMallocManaged(&room,  sizeof(Room));
	room->RoomCopy(*m_room);
}

automatedLayout::automatedLayout(vector<float>in_weights) {

	// constrains = new layoutConstrains(m_room);
	min_cost = INFINITY;

	// cout<<"shenmemaobing: " <<room->deviceObjs[1].id<<endl;
	hipMallocManaged(&weights, in_weights.size() * sizeof(float));
	for(int i=0;i<in_weights.size();i++)
		weights[i] = in_weights[i];

	hipMallocManaged(&resTransAndRot,  4 * sizeof(float));
	// for(int i=0;i<RES_NUM*4;i++)
	// 	resTransAndRot[i] = i;
	// float tmpf[] = {1.0f, 1.5f, 0.5f,1.0f};
	// hipMemcpy(resTransAndRot, tmpf, 4*sizeof(float), hipMemcpyHostToDevice);
}


void automatedLayout:: generate_suggestions(){
	if(room->objctNum == 0)
		return;
    int * pickedIdxs; //should be in global mem


    hipMallocManaged(&pickedIdxs, nBlocks * nTimes * sizeof(int));
    for(int i=0; i<nBlocks*nTimes; i++)
        pickedIdxs[i] = rand()%room->objctNum;

	//memory to store result, should be in global mem


    //dynamic shared mem, <<<nb, nt, sm>>>
	//obj +  temparing + room + weight
	// int sharedMem = nBlocks * (sizeof(room->objects)+ 2*sizeof(float)+ sizeof(*room) + sizeof(*deviceWeights));
	int objMem = nBlocks * room->objctNum * sizeof(singleObj);
	int temMem = nBlocks * sizeof(float);
	float * rArray;
	hipMallocManaged(&rArray, nBlocks * sizeof(float));

	AssignFurnitures<<<nBlocks, room->objctNum, objMem >>>(pickedIdxs, time(NULL));
	hipDeviceSynchronize();
	Do_Metropolis_Hastings<<<nBlocks, room->objctNum, temMem>>>(time(NULL), rArray);
	hipDeviceSynchronize();

	hipFree(resTransAndRot);
	hipFree(pickedIdxs);
	hipFree(weights);
	hipFree(room);


    for(int i=0;i<nBlocks;i++){
        // for(int j=0; j<numofObjs; j++)
            cout<<rArray[i]<<" ";
        cout<<endl;
    }

}
void automatedLayout::random_along_wall(int furnitureID){

}
float automatedLayout::cost_function(){
	return 0;
}

// void automatedLayout::initial_assignment(){
// 	for (int i = 0; i < room->freeObjIds.size(); i++) {
// 		singleObj* obj = &room->objects[room->freeObjIds[i]];
// 		if (obj->adjoinWall)
// 			random_along_wall(room->freeObjIds[i]);
// 		else if (obj->alignedTheWall)
// 			room->set_obj_zrotation(room->walls[rand() % room->wallNum].zrotation, room->freeObjIds[i]);
// 	}
// 	room->update_furniture_mask();
// 	min_cost = cost_function();
// 	if (min_cost == -1)
// 		min_cost = INFINITY;
//
// }
// int main(int argc, char** argv){
//     setUpDevices();
//     seed = time(NULL);
//     srand(seed);
//     generate_suggestions();
//     return 0;
// }
