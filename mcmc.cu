#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include<fstream>
#include <limits.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
// #include "predefinedConstrains.h"
#include "room.cuh"
#include <time.h>
#define RES_NUM 1
#define THREADHOLD_T 0.8
#define MAX_THREAD_NUM 64
using namespace std;

const unsigned int nBlocks = 3;
const unsigned int nThreads = 16;
const unsigned int WHICH_GPU = 0;
const unsigned int nTimes = 1;

struct sharedWrapper;
extern __shared__ sharedWrapper sWrapper[];

__device__ __managed__ float weights[11]={1.0f};
__device__ __managed__ float resTransAndRot[RES_NUM * 4];
struct sharedWrapper{
    sharedRoom *wRoom;
    singleObj *wObjs;
    unsigned char *wMask;
    float *wFloats;
};
void setUpDevices(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(WHICH_GPU <= deviceCount) {
    hipError_t err = hipSetDevice(WHICH_GPU);
    if(err != hipSuccess)
        cout<< "CUDA error:" <<hipGetErrorString(err)<<endl;
    }
    else {
        cout << "Invalid GPU device " << WHICH_GPU << endl;
        exit(-1);
    }
    int wgpu;
    hipGetDevice(&wgpu);
    hipDeviceReset();
}

__device__
float density_function(float beta, float cost) {
    // printf("%f-%f\n", beta, cost);
	return exp2f(-beta * cost);
}

__device__
float get_randomNum(unsigned int seed, int maxLimit) {
  hiprandState_t state;
  //seed, sequence number(multiple cores), offset
  hiprand_init(seed, 0,0, &state);
  return hiprand(&state) % maxLimit;
}


__device__
void changeTemparature(float * temparature, unsigned int seed){
    int t1 = int(get_randomNum(blockIdx.x, nBlocks+1))%nBlocks;
    int t2 = t1;
    int times = 0;
    while(t2 == t1 && times++ < 3){
        t2 = int(get_randomNum(blockIdx.x, nBlocks+1))%nBlocks;
    }
    if(t2 == t1)
        t2 = (t1+1)%nBlocks;
    float tmp = temparature[t1];
    temparature[t1] = temparature[t2];
    temparature[t2] = tmp;
}
__device__
void randomly_perturb(){

}
__device__
void restoreOrigin(){

}
__device__
void getTemporalTransAndRot(){

}
__device__ float t(float d, float m, float M, int a = 2){
    if (d < m)
		return powf((d / m), float(a));
	else if (d > M)
		return powf((M / d), float(a));
	else
		return 1.0f;
}

//TODO:
__device__
int get_sum_furnitureMsk(unsigned char* mask){
    //return furnitureMsk by different blockIdx
    return 100*(blockIdx.x + 1);
}
//TODO:
//void get_all_reflection(map<int, Vec3f> focalPoint_map, vector<Vec3f> &reflectTranslate, vector<float> & reflectZrot, float refk= INFINITY);
__device__
void get_pairwise_relation(const singleObj& obj1, const singleObj& obj2, int&pfg, float&m, float&M, int & wallRelId){

}
//Clearance :
//Mcv(I) that minimize the overlap between furniture(with space)
__device__
void cal_clearance_violation(float& mcv){
    float overlappingArea = get_sum_furnitureMsk(&sWrapper[0].wMask[blockIdx.x * sWrapper[0].wRoom->mskCount ]) - sWrapper[0].wRoom->obstacleArea - sWrapper[0].wRoom->wallArea;
    mcv = sWrapper[0].wRoom->indepenFurArea - overlappingArea;
    mcv = (mcv < 0)? 0 : mcv;
}
//Circulation:
//Mci support circulation through the room and access to all of the furniture.
__device__
void cal_circulation_term(float& mci){
    mci = 0;
}
//Pairwise relationships:
//Mpd: for example  coffee table and seat
//mpa: relative direction constraints
__device__ void cal_pairwise_relationship(float& mpd, float& mpa){

}
//Conversation
//Mcd:group a collection of furniture items into a conversation area
__device__ void cal_conversation_term(float& mcd, float& mca){}
//balance:
//place the mean of the distribution of visual weight at the center of the composition
__device__ void cal_balance_term(float &mvb){}
//Alignment:
//compute furniture alignment term
__device__ void cal_alignment_term(float& mfa, float&mwa){}
//Emphasis:
//compute focal center
__device__ void cal_emphasis_term(float& mef, float& msy, float gamma = 1){}
__device__
void get_constrainTerms(float* costList, int weightTerm){
	switch (weightTerm) {
		case 0://mcv
			cal_clearance_violation(costList[threadIdx.x]);
			break;
		case 1://Mci
			cal_circulation_term(costList[threadIdx.x]);
			break;
		case 2:
			cal_pairwise_relationship(costList[threadIdx.x], costList[threadIdx.x + 1]);
			break;
		case 3:
			cal_conversation_term(costList[threadIdx.x+1], costList[threadIdx.x+2]);
			break;
		case 4:
			cal_balance_term(costList[threadIdx.x+2]);
			break;
		case 5:
			if(sWrapper[0].wRoom->wallNum != 0)
				cal_alignment_term(costList[threadIdx.x+2], costList[threadIdx.x+3]);
			break;
		case 6:
			cal_emphasis_term(costList[threadIdx.x+3],costList[threadIdx.x+4]);
			break;
		default:
			break;
	}
}

__device__
float getWeightedCost(float* costList, int consStartId){
    if(threadIdx.x >= consStartId){
        get_constrainTerms(costList, threadIdx.x-consStartId);
        costList[threadIdx.x] = threadIdx.x;
    }

    //else do nothing, empty the first #numofObjs slots
    __syncthreads();
    float res = 0;
    for(int i=0; i<WEIGHT_NUM; i++)
        res += weights[i] * costList[consStartId + i];
    return res;
}

__device__
void Metropolis_Hastings(float* costList, float* temparature, int* pickedIdxs, unsigned int seed){
    float cpost, p0, p1, alpha;
    int startId = blockIdx.x * nThreads;
    int index = startId + threadIdx.x;
    costList[index] = 0;
    float cpre = 0;//getWeightedCost(&costList[startId], sWrapper[0].wRoom->objctNum);
    //first thread cost is the best cost of block
    costList[startId] = cpre;
    for(int nt = 0; nt<nTimes; nt++){
        if(pickedIdxs[blockIdx.x] == threadIdx.x){
            if(nt % 10 == 0)
                changeTemparature(temparature, seed+blockIdx.x);
            p0 = density_function(temparature[blockIdx.x], cpre);
            randomly_perturb(/*original keep sth to restore*/);
        }
        __syncthreads();

        cpost = getWeightedCost(&costList[startId], sWrapper[0].wRoom->objctNum);
        costList[index] = 0;
        if(pickedIdxs[blockIdx.x] == threadIdx.x){
            p1 = density_function(temparature[blockIdx.x], cpost);
            alpha = fminf(1.0f, p1/p0);
            if(alpha > THREADHOLD_T)
                restoreOrigin();
            else if(cpost < costList[blockIdx.x]){
                getTemporalTransAndRot();
                costList[startId] = cpost;
                cpre = cpost;
            }
            pickedIdxs[blockIdx.x] = int(get_randomNum(seed+blockIdx.x, sWrapper[0].wRoom->objctNum));
        }
        __syncthreads();
    }
}

__global__
void Do_Metropolis_Hastings(sharedWrapper *gWrapper, unsigned int seed){
    sWrapper[0] = *gWrapper;
    if(blockIdx.x !=0 ){
        if(threadIdx.x < sWrapper[0].wRoom->objctNum){
            int objId = blockIdx.x * sWrapper[0].wRoom->objctNum + threadIdx.x;
            sWrapper[0].wObjs[objId] = sWrapper[0].wObjs[threadIdx.x];
        }
        //can be optimized
        if(threadIdx.x == 0){
            for(int i=0; i<sWrapper[0].wRoom->mskCount; i++)
                sWrapper[0].wMask[blockIdx.x *sWrapper[0].wRoom->mskCount + i] = sWrapper[0].wMask[i];
        }
    }
    float* costList = sWrapper[0].wFloats;
    float* temparature = (float *) & costList[nBlocks * nThreads];
    int* pickedIdxs = (int *)& temparature[nBlocks];
	temparature[blockIdx.x] = -get_randomNum(seed+blockIdx.x, 100) / 10;
    pickedIdxs[blockIdx.x] = int(get_randomNum(seed+blockIdx.x, sWrapper[0].wRoom->objctNum));
    // printf("%d\n", pickedIdxs[blockIdx.x]);
    Metropolis_Hastings(costList, temparature, pickedIdxs, seed);
    __syncthreads();
}



void generate_suggestions(Room * m_room){
    sharedWrapper *gWrapper;
    hipMallocManaged(&gWrapper,  sizeof(sharedWrapper));

    hipMallocManaged(&gWrapper->wRoom, sizeof(sharedRoom));
    m_room->CopyToSharedRoom(gWrapper->wRoom);

    int objMem = nBlocks *m_room->objctNum * sizeof(singleObj);
    hipMallocManaged(&gWrapper->wObjs, objMem);
	for(int i=0; i<m_room->objctNum; i++)
		gWrapper->wObjs[i] = m_room->objects[i];

    int tMem = m_room->colCount * m_room->rowCount * sizeof(unsigned char);
    hipMallocManaged(&gWrapper->wMask, nBlocks *tMem);
    hipMemcpy(gWrapper->wMask, m_room->furnitureMask, tMem, hipMemcpyHostToDevice);

	int floatMem =  nBlocks *(2+nThreads) * sizeof(float);
    hipMallocManaged(&gWrapper->wFloats, floatMem);

	Do_Metropolis_Hastings<<<nBlocks, nThreads, sizeof(*gWrapper)>>>(gWrapper, time(NULL));
	hipDeviceSynchronize();

    hipFree(gWrapper->wRoom);
    hipFree(gWrapper->wObjs);
    hipFree(gWrapper->wMask);
    hipFree(gWrapper->wFloats);
    hipFree(gWrapper);
}

__device__ __host__
void random_along_wall(int furnitureID) {
}


// void initial_assignment(){
//     for (int i = 0; i < room->freeObjNum; i++) {
//     	singleObj* obj = &room->deviceObjs[room->freeObjIds[i]];
//     	if (obj->adjoinWall)
//     		random_along_wall(room->freeObjIds[i]);
//     	else if (obj->alignedTheWall)
//     		room->set_obj_zrotation(&room->deviceObjs[room->freeObjIds[i]], room->deviceWalls[rand() % room->wallNum].zrotation);
//     }
//     room->update_furniture_mask();
// }

void startToProcess(Room * m_room){
    if(m_room->objctNum == 0)
        return;
	setUpDevices();

    clock_t start, finish;
    float costtime;
    start = clock();

	generate_suggestions(m_room);

    finish = clock();
    costtime = (float)(finish - start) / CLOCKS_PER_SEC;
    cout<<"Runtime: "<<costtime<<endl;
}
void setupDebugRoom(Room* room){
    float wallParam1[] = {-200, 150, 200, 150};
    float wallParam2[] = {-200, -150, 200, -150};
    float objParam[] = {0, 0, 100, 50, 0, 0, 10};
    float fpParam[] = {0, 150, 0};
    float mWeights[] = {1, 1.0, 3.0, 2.0, 1.0, 1.0, 1.0, 3.0, 3.0, 1.0, 0.5};

    room->initialize_room();
    room->add_a_wall(vector<float>(wallParam1,wallParam1 + 4));
    room->add_a_wall(vector<float>(wallParam2,wallParam2 + 4));
    room->add_an_object(vector<float>(objParam,objParam + 7));
    room->add_a_focal_point(vector<float>(fpParam,fpParam + 3));

    for(int i=0;i<11;i++)
        weights[i] = mWeights[i];
}
void parser_inputfile(const char* filename, Room * parser_inputfile) {
	ifstream instream(filename);
	string str;
	vector<vector<float>> parameters;
	vector<char> cateType;
	char  delims[] = " :,\t\n";
	char* context = nullptr;
	while (instream && getline(instream, str)) {
		if (!str.length())
			continue;
		char * charline = new char[300];
		int r = strcpy_s(charline, 300, str.c_str());
		char * itemCate = strtok_s(charline,delims,&context);
		vector<float>param;
		char * token = strtok_s(nullptr, delims, &context);
		while (token != nullptr) {
			param.push_back(atof(token));
			token = strtok_s(nullptr, delims, &context);
		}
		parameters.push_back(param);
		cateType.push_back(itemCate[0]);
	}
	instream.close();
	int itemNum = cateType.size();
	vector<vector<float>> fixedObjParams;
	vector<vector<float>> mergedObjParams;
	vector<int> groupedIds;
	int startId = 0;
	if (cateType[0] == 'r') {
		parser_inputfile->initialize_room(parameters[0][0], parameters[0][1]);
		startId = 1;
	}
	else if(!parser_inputfile->initialized)
		parser_inputfile->initialize_room();
	for (int i = startId; i < itemNum; i++) {
		switch (cateType[i])
		{
		case '#':
			break;
		//add a new wall
		case 'w':
			parser_inputfile->add_a_wall(parameters[i]);
			break;
		case 'f':
			parser_inputfile->add_an_object(parameters[i]);
			break;
		case 'p':
			parser_inputfile->add_a_focal_point(parameters[i]);
			break;
		case 'v':
            for(int k=0;k<parameters[i].size(); k++)
                weights[k] = parameters[i][k];
			break;
        default:
            break;
        }
    }
}
int main(int argc, char** argv){
    char* filename;
    /*if (argc < 2) {
        filename = new char[9];
        strcpy(filename, "input.txt");
    }
    else
        filename = argv[1];*/
	char* existance_file;
	filename = new char[100];
	existance_file = new char[100];
	int r = strcpy_s(filename, 100, "E:/layoutParam.txt");
	r = strcpy_s(existance_file, 100, "E:/fixedObj.txt");
	Room* parserRoom = new Room();
    setupDebugRoom(parserRoom);
	// parser_inputfile(filename, parserRoom);
	// parser_inputfile(existance_file, room, weights);
	// if (parserRoom != nullptr && (parserRoom->objctNum != 0 || parserRoom->wallNum != 0))
    startToProcess(parserRoom);
	return 0;
}
