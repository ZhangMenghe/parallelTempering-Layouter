#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include<fstream>
#include <limits.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
// #include "predefinedConstrains.h"
#include "room.cuh"
#define RES_NUM 1

using namespace std;
// using namespace cv;

const unsigned int nBlocks = 10 ;
const unsigned int WHICH_GPU = 0;
const unsigned int nTimes =20;

void roomInitialization(Room* m_room);
void generate_suggestions();
extern __shared__ singleObj sObjs[];
extern __shared__ float sFloats[];
__device__ __managed__ Room* room;
__device__ __managed__ float weights[11]={1.0f};
__device__ __managed__ float resTransAndRot[RES_NUM * 4];

void setUpDevices(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(WHICH_GPU <= deviceCount) {
    hipError_t err = hipSetDevice(WHICH_GPU);
    if(err != hipSuccess)
        cout<< "CUDA error:" <<hipGetErrorString(err)<<endl;
    }
    else {
        cout << "Invalid GPU device " << WHICH_GPU << endl;
        exit(-1);
    }
    int wgpu;
    hipGetDevice(&wgpu);
    hipDeviceReset();
}
void debugCaller(){
    room->set_obj_zrotation(&room->deviceObjs[0], PI);
    room->set_obj_translation(&room->deviceObjs[0], -50, 0);
    // room->get_nearest_wall_dist(&room->deviceObjs[0]);
}
void startToProcess(Room * m_room){
	//cout<<"hello from mcmc"<<endl;
	setUpDevices();
	roomInitialization(m_room);

	// layout->initial_assignment(m_room);
    debugCaller();
	generate_suggestions();
   // 	// layout->display_suggestions();
}

__device__
float density_function(float beta, float cost) {
    // printf("%f-%f\n", beta, cost);
	return exp2f(-beta * cost);
}
__device__
void cost_function(){}
__device__
float get_randomNum(unsigned int seed, int maxLimit) {
  /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t state;

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);

  /* hiprand works like rand - except that it takes a state as a parameter */
  return hiprand(&state) % maxLimit;
 // int res = hiprand(&state) % maxLimit;
 // printf("%d ", res);
 // return res;
}


__device__
void changeTemparature(float * temparature, unsigned int seed){
    int t1 = get_randomNum(seed, nBlocks);
    int t2=t1;
    while(t2 == t1)
        t2 = get_randomNum(seed + 100, nBlocks);
    float tmp = temparature[t1];
    temparature[t1] = temparature[t2];
    temparature[t2] = tmp;
}

__device__
void Metropolis_Hastings(int* pickedIdAddr, float* costList, float* temparature, unsigned int seed){

}
__global__
void Do_Metropolis_Hastings(int * pickedIdxs, unsigned int seed){
    float* constrainParams = sFloats;
	float* costList = (float *) & constrainParams[nBlocks * 11];
    float* temparature = (float *) & costList[nBlocks];

	temparature[blockIdx.x] = -get_randomNum(seed+blockIdx.x, 100) / 10;
	int* pickedIdAddr = &pickedIdxs[blockIdx.x * nTimes];
    // Metropolis_Hastings(pickedIdAddr, costList, temparature, seed);
	__syncthreads();

}
__global__
void AssignFurnitures(){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	sObjs[index] = room->deviceObjs[threadIdx.x];
	__syncthreads();
}



void roomInitialization(Room* m_room){
	hipMallocManaged(&room,  sizeof(Room));
	room->RoomCopy(*m_room);
}




void generate_suggestions(){
	if(room->objctNum == 0)
		return;
    int * pickedIdxs; //should be in global mem


    hipMallocManaged(&pickedIdxs, nBlocks * nTimes * sizeof(int));
    //block1.....block2....
    for(int i=0; i<nBlocks*nTimes; i++)
        pickedIdxs[i] = rand()%room->objctNum;

    //dynamic shared mem, <<<nb, nt, sm>>>
	//obj +  temparing + room + weight
	// int sharedMem = nBlocks * (sizeof(room->objects)+ 2*sizeof(float)+ sizeof(*room) + sizeof(*deviceWeights));
	int objMem = nBlocks * room->objctNum * sizeof(singleObj);
	int floatMem = 13 * nBlocks * sizeof(float);

	AssignFurnitures<<<nBlocks, room->objctNum, objMem>>>();
	hipDeviceSynchronize();
	Do_Metropolis_Hastings<<<nBlocks, room->objctNum, floatMem>>>(pickedIdxs, time(NULL));
	hipDeviceSynchronize();

    room->freeMem();
	hipFree(pickedIdxs);
	hipFree(room);


    // for(int i=0;i<nBlocks;i++){
    //     // for(int j=0; j<numofObjs; j++)
    //         cout<<rArray[i]<<" ";
    //     cout<<endl;
    // }

}
__device__ __host__
void random_along_wall(int furnitureID) {
}


void initial_assignment(){
    for (int i = 0; i < room->freeObjNum; i++) {
    	singleObj* obj = &room->deviceObjs[room->freeObjIds[i]];
    	if (obj->adjoinWall)
    		random_along_wall(room->freeObjIds[i]);
    	else if (obj->alignedTheWall)
    		room->set_obj_zrotation(&room->deviceObjs[room->freeObjIds[i]], room->deviceWalls[rand() % room->wallNum].zrotation);
    }
    room->update_furniture_mask();
}


void parser_inputfile(const char* filename, Room * parser_inputfile) {
	ifstream instream(filename);
	string str;
	vector<vector<float>> parameters;
	vector<char> cateType;
	char  delims[] = " :,\t\n";
	char* context = nullptr;
	while (instream && getline(instream, str)) {
		if (!str.length())
			continue;
		char * charline = new char[300];
		int r = strcpy_s(charline, 300, str.c_str());
		char * itemCate = strtok_s(charline,delims,&context);
		vector<float>param;
		char * token = strtok_s(nullptr, delims, &context);
		while (token != nullptr) {
			param.push_back(atof(token));
			token = strtok_s(nullptr, delims, &context);
		}
		parameters.push_back(param);
		cateType.push_back(itemCate[0]);
	}
	instream.close();
	int itemNum = cateType.size();
	vector<vector<float>> fixedObjParams;
	vector<vector<float>> mergedObjParams;
	vector<int> groupedIds;
	int startId = 0;
	if (cateType[0] == 'r') {
		parser_inputfile->initialize_room(parameters[0][0], parameters[0][1]);
		startId = 1;
	}
	else if(!parser_inputfile->initialized)
		parser_inputfile->initialize_room();
	for (int i = startId; i < itemNum; i++) {
		switch (cateType[i])
		{
		case '#':
			break;
		//add a new wall
		case 'w':
			parser_inputfile->add_a_wall(parameters[i]);
			break;
		case 'f':
			parser_inputfile->add_an_object(parameters[i]);
			break;
		case 'p':
			parser_inputfile->add_a_focal_point(parameters[i]);
			break;
		case 'v':
            for(int k=0;k<parameters[i].size(); k++)
                weights[k] = parameters[i][k];
			break;
        default:
            break;
        }
    }
}
int main(int argc, char** argv){
    char* filename;
    /*if (argc < 2) {
        filename = new char[9];
        strcpy(filename, "input.txt");
    }
    else
        filename = argv[1];*/
	char* existance_file;
	filename = new char[100];
	existance_file = new char[100];
	int r = strcpy_s(filename, 100, "E:/layoutParam.txt");
	r = strcpy_s(existance_file, 100, "E:/fixedObj.txt");
	Room* parserRoom = new Room();
	parser_inputfile(filename, parserRoom);
	// parser_inputfile(existance_file, room, weights);
	if (parserRoom != nullptr && (parserRoom->objctNum != 0 || parserRoom->wallNum != 0))
        startToProcess(parserRoom);
	return 0;
}
