#include "hip/hip_runtime.h"
#include <iostream>
#include "room.cuh"
#include "cudaroom.cuh"
#include "utils.cuh"
#include "constrainTerms.cuh"
#include "hostUtils.h"
using namespace std;

#define THREADHOLD_T 0.7

const unsigned int nBlocks = 40;
const unsigned int nThreads = 32;//it's werid
const unsigned int WHICH_GPU = 0;

struct sharedWrapper;
extern __shared__ sharedWrapper sWrapper[];

struct sharedWrapper{
    int nTimes;
    sharedRoom *wRoom;//1
    singleObj *wObjs;//nblocks
    unsigned char *initialMask;//1
    unsigned char *wMask;//nblocks
    unsigned char* backMask;//nblocks
    float * obstacleVertices;//1
    float * wmaskArea;//nblocks * 2
    float *wFloats;//1
    int *wPairRelation;//1
    float * resTransAndRot;//1 for all objs and all blocks
};

void setUpDevices(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(WHICH_GPU <= deviceCount) {
    hipError_t err = hipSetDevice(WHICH_GPU);
    if(err != hipSuccess)
        cout<< "CUDA error:" <<hipGetErrorString(err)<<endl;
    }
    else {
        cout << "Invalid GPU device " << WHICH_GPU << endl;
        exit(-1);
    }
    int wgpu;
    hipGetDevice(&wgpu);
    hipDeviceReset();
}

__device__
void changeTemparature(float * temparature){
    int t1 = get_int_random(nBlocks);
    int t2 = t1;
    while(t2 == t1)
        t2 = get_int_random(nBlocks);
    float tmp = temparature[t1];
    temparature[t1] = temparature[t2];
    temparature[t2] = tmp;
}

__device__
void random_along_wall(sharedRoom * room, singleObj * obj){
    wall * swall = &room->deviceWalls[get_int_random(room->wallNum)];
    float mwidth, mheight;
    if(get_int_random(2)==0){
        mwidth = obj->objWidth; mheight = obj->objHeight;
        set_obj_zrotation(obj, swall->zrotation);
    }else{
        mwidth = obj->objHeight; mheight = obj->objWidth;
        set_obj_zrotation(obj, PI/2-swall->zrotation);
    }

    float width_ran = swall->width - mwidth, height_ran =swall->width-mheight;
    float rh, rw;
    int mp = (swall->translation[0] >0 || swall->translation[1]>0)? -1:1;
    if(fabsf(swall->b) < 0.01){
        rh = min(swall->vertices[1], swall->vertices[3]) + get_float_random(height_ran) + obj->boundingBox.height/2;
        set_obj_translation(room, obj, swall->translation[0] + mp*(mwidth/2+0.01), rh);
    }
    else if(fabsf(swall->a) < 0.01){
        rw = min(swall->vertices[0], swall->vertices[2]) + get_float_random(width_ran) + obj->boundingBox.width/2;
        set_obj_translation(room, obj, rw,swall->translation[1] + mp*(mheight/2+0.01) );
    }
    else{
        //TODO:
        printf("CANNOT ACCEPT OBLIQUE WALL\n");
    }
}



__device__
void initial_assignment(sharedRoom* room, singleObj * objs,
                        unsigned char* initialMask, unsigned char * mask, unsigned char * backupMask,
                         float* tmpSlot){
    if(threadIdx.x < room->objctNum){
        singleObj * obj = &objs[threadIdx.x];
        if (obj->adjoinWall)
            random_along_wall(room, obj);

        else if (obj->alignedTheWall)
            set_obj_zrotation(obj, room->deviceWalls[get_int_random(room->wallNum)].zrotation);

        //INITIALIZE COST
        int singleSize = room->objctNum * 4 + 1;
        for(int i=0; i<MAX_KEPT_RES; i++){
            sWrapper[0].resTransAndRot[singleSize*i + threadIdx.x * 4] = INFINITY;
        }
    }
    memcpy(mask, initialMask, room->mskCount*sizeof(unsigned char));

    //all threads to do update masks
    for (int i = 0; i < room->objctNum; i++) {
        mRect2f rect = get_circulate_boundingbox(room, &objs[i].boundingBox);
        update_mask_by_boundingBox(backupMask, rect, room->rowCount/2, room->colCount, threadIdx.x, nThreads);
        if(!objs[i].isFixed){
            draw_objMask_patch(room, &objs[i], tmpSlot, threadIdx.x, nThreads);
            draw_patch_on_union_mask(mask, &objs[i], room->rowCount/2, room->colCount, threadIdx.x, nThreads);
        }
    }


    __syncthreads();

    sumUpMask(room, mask, tmpSlot, &sWrapper[0].wmaskArea[2*blockIdx.x], nThreads);

    sumUpMask(room, backupMask, tmpSlot, &sWrapper[0].wmaskArea[2*blockIdx.x+1], nThreads);
}

__device__
void getTemporalTransAndRot(sharedRoom * room, singleObj* objs, float * results, float cost){
    float maxCost = results[0];
    int i = 1, maxPos = 0, singleSize = room->objctNum * 4 + 1;
    for(i=1; i<MAX_KEPT_RES; i++){
        if(maxCost == INFINITY)
            break;
        if(results[singleSize * i] >maxCost){
            maxPos=i; maxCost = results[singleSize * i];
        }
    }
    if(cost < maxCost){
        int baseId = singleSize * maxPos;
        results[baseId] = cost;
        for(int i=0; i<room->objctNum; i++){
            results[baseId + 4*i + 1] = objs[i].translation[0];
            results[baseId + 4*i + 2] = objs[i].translation[1];
            results[baseId + 4*i + 3] = objs[i].translation[2];
            results[baseId + 4*i + 4] = objs[i].zrotation;
        }
    }
}

__device__
int randomly_perturb(sharedRoom* room, singleObj * objs, int pickedIdx,
                    unsigned char*initialMask, unsigned char * mask, unsigned char* backupMask,
                    float* tmpSlot){
    int secondChangeId = -1;
    singleObj * obj = &objs[pickedIdx];
    storeOrigin(obj);
    int index = blockIdx.x * nThreads + threadIdx.x;
    // REAL RANDOM HERE
    if(threadIdx.x == 0){
        if (obj->adjoinWall)
            random_along_wall(room, obj);
        else{
            int trytimes = 0;
            int randomMethod = (room->objctNum < 2 || obj->alignedTheWall)? 2: 3;
            switch (get_int_random(randomMethod, index)){
                // randomly rotate
                case 0:
                    if (obj->alignedTheWall)
                        set_obj_zrotation(obj, room->deviceWalls[get_int_random(room->wallNum, index)].zrotation);
                    else
                        set_obj_zrotation(obj, get_float_random(PI, index));
                    break;
                case 1:
                    while(set_obj_translation(room, obj,
                                            get_float_random(room->half_width, index),
                                            get_float_random(room->half_height, index),true));
                    break;
                case 2:
                    singleObj * obj2;

                    // float tmpx = obj->translation[0], tmpy=obj->translation[1], tmprot = obj->zrotation;
                    while(trytimes++ < 5){
                        obj2 = &objs[get_int_random(room->objctNum, index)];
                        if(obj2->id == pickedIdx || obj2->adjoinWall || obj2->alignedTheWall)
                            continue;
                        storeOrigin(obj2);

                        if(!set_obj_translation(room, obj, obj2->translation[0], obj2->translation[1]))
                            continue;
                        if(!set_obj_translation(room, obj2, obj->lastTransAndRot[0], obj->lastTransAndRot[1])){
                            set_obj_translation(room, obj, obj->lastTransAndRot[0], obj->lastTransAndRot[1]);
                            continue;
                        }
                        break;
                    }
                    if(trytimes >= 5)
                        while(set_obj_translation(room, obj,
                                                get_float_random(room->half_width, index),
                                                get_float_random(room->half_height, index),true));
                    else{
                        set_obj_zrotation(obj, obj2->zrotation);
                        set_obj_zrotation(obj2, obj->lastTransAndRot[3]);
                        secondChangeId = obj2->id;
                    }
                    break;
                default:
                    break;
                }//end switch
        }// end not adjoint wall
    }//end thread == 0

    draw_objMask_patch(room, obj, tmpSlot, threadIdx.x, nThreads);
    change_an_obj_backupMask(room, obj, backupMask, nThreads);
     if(secondChangeId!=-1){
         draw_objMask_patch(room, &objs[secondChangeId], tmpSlot, threadIdx.x, nThreads);
         change_an_obj_backupMask(room,  &objs[secondChangeId], backupMask, nThreads);
     }


    // memset(mask, 0, room->mskCount * sizeof(unsigned char));
    memcpy(mask, initialMask,room->mskCount*sizeof(unsigned char) );
    for (int i = 0; i < room->freeObjNum; i++)
        draw_patch_on_union_mask(mask, &objs[room->freeObjIds[i]], room->rowCount/2, room->colCount, threadIdx.x, nThreads);

    __syncthreads();

    sumUpMask(room, mask, tmpSlot, &sWrapper[0].wmaskArea[2*blockIdx.x], nThreads);
    sumUpMask(room, backupMask, tmpSlot, &sWrapper[0].wmaskArea[2*blockIdx.x+1], nThreads);

    return secondChangeId;
}

__device__
void Metropolis_Hastings(float* costList, float* temparature, int*pickedupIds){
    float cpost, p0, p1, alpha;
    sharedRoom * room = sWrapper[0].wRoom;
    singleObj * objsBlock = &sWrapper[0].wObjs[blockIdx.x * room->objctNum];
    int startId = blockIdx.x * nThreads;
    int index = startId + threadIdx.x;
    int maskStart = room->mskCount * blockIdx.x;
    int secondChangeId,pickedId;

    //sharedRoom* room, singleObj * objs,  unsigned char * mask, float* tmpSlot, int threadStride)
    initial_assignment(room, objsBlock,
                        sWrapper[0].initialMask, &sWrapper[0].wMask[maskStart], &sWrapper[0].backMask[maskStart],
                        &costList[startId]);

    getWeightedCost(room, objsBlock, sWrapper[0].wPairRelation, &sWrapper[0].wmaskArea[2*blockIdx.x], &costList[startId]);
    __syncthreads();

    float cpre = sumUp_weighted_dataInShare(&costList[startId+1], weights, WEIGHT_NUM);
    getTemporalTransAndRot(room, objsBlock, sWrapper[0].resTransAndRot, cpre);
    if(blockIdx.x ==0 && threadIdx.x == 0)
        displayResult(costList, weights);
    if(sWrapper[0].nTimes == 0)
        return;
    for(int nt = 0; nt<sWrapper[0].nTimes; nt++){
        if(threadIdx.x == 0){
            if(nBlocks>1 && nt % 10 == 0)
                changeTemparature(temparature);
            p0 = density_function(temparature[blockIdx.x], cpre);
        }

        pickedId = pickedupIds[blockIdx.x];

        // if(threadIdx.x ==0)
            // printf("block: %d pickup: %d\n",blockIdx.x, pickedId );
            // fprintf( stderr,"threadIdx: %d, nTimes: %d\n", threadIdx.x, nt);
            //printf("threadIdx: %d, nTimes: %d\n", threadIdx.x, nt);
        __syncthreads();

        if(threadIdx.x == 0){
            pickedupIds[blockIdx.x] = room->freeObjIds[get_int_random(room->freeObjNum)];
            // printf("block: %d pickup: %d\n",blockIdx.x, pickedId );
        }


        secondChangeId = randomly_perturb(room, objsBlock, pickedId,
                        sWrapper[0].initialMask, &sWrapper[0].wMask[maskStart], &sWrapper[0].backMask[maskStart],
                        &costList[startId]);

        getWeightedCost(room, objsBlock, sWrapper[0].wPairRelation, &sWrapper[0].wmaskArea[2*blockIdx.x], &costList[startId]);
        // if(threadIdx.x == 0 && nt%10==0 ){
        //     for(int i=0; i<2; i++)
        //         printf("obj: %d, loc: %f, %f\n",i, objsBlock[i].translation[0], objsBlock[i].translation[1] );
            // displayResult(costList, weights);
        // }

        __syncthreads();

        cpost = sumUp_weighted_dataInShare(&costList[startId+1], weights, WEIGHT_NUM);

        costList[index] = 0;

        if(threadIdx.x == 0){
            p1 = density_function(temparature[blockIdx.x], cpost);
            alpha = fminf(1.0f, p1/p0);
            // printf("alpha: %f cpre: %f cpost: %f\n",alpha, cpre, cpost );
            if(alpha < THREADHOLD_T){
                restoreOrigin(room, &sWrapper[0].wMask[maskStart],&costList[startId],
                                &objsBlock[pickedId], nThreads);
                if(secondChangeId!=-1)
                    restoreOrigin(room, &sWrapper[0].wMask[maskStart],&costList[startId],
                                &objsBlock[secondChangeId], nThreads);
            }
            else if(cpost < cpre){
                getTemporalTransAndRot(room, objsBlock, sWrapper[0].resTransAndRot, cpost);
                cpre = cpost;
            }
        }//end thread 0
    }//end for
}
__device__
void Initialize_Room_In_Device(sharedRoom* room, singleObj* objs, unsigned char* initialMask,float*obsVertices,float*tmpSlot){
    for(int k=0; k<room->obstacleNum; k++){
        float minx= obsVertices[0],maxx = obsVertices[0], miny=obsVertices[1],maxy = obsVertices[1];
        for(int i=1; i<4; i++){
            minx = (obsVertices[2*i] < minx)? obsVertices[2*i]:minx;
            maxx = (obsVertices[2*i] > maxx)? obsVertices[2*i]:maxx;
            miny = (obsVertices[2*i + 1] < miny)? obsVertices[2*i+1]:miny;
            maxy = (obsVertices[2*i + 1] > maxy)? obsVertices[2*i+1]:maxy;
        }
        mRect2f boundingBox = {minx, maxy, maxx-minx, maxy-miny};

        update_mask_by_object(initialMask, tmpSlot, &obsVertices[8*k], boundingBox,
                              room->rowCount/2, room->colCount,
                              threadIdx.x, nThreads);
    }
    __syncthreads();
    sumUpMask(room, initialMask, tmpSlot, &room->obstacleArea, nThreads);
    __syncthreads();

    for(int i=0; i<room->objctNum; i++){
        if(objs[i].isFixed){
            draw_objMask_patch(room, &objs[i], tmpSlot, threadIdx.x, nThreads);
            draw_patch_on_union_mask(initialMask, &objs[i], room->rowCount/2, room->colCount, threadIdx.x, nThreads);
        }
    }

    //tackle with oblique wall
    /*for(int i=0; i<room->wallNum; i++){
        wall* cwall = &room->deviceWalls[i];
        if(fabs(fmod(cwall->zrotation, 90)) > 0.01){
            printf("wall: %d\n", cwall->id);
            float ax, ay, bx, by;
            if(fabs(wall->vertices[0]) > fabs(cwall->vertices[2])){
                ax = cwall->vertices[0]; ay = cwall->vertices[1];
                bx = cwall->vertices[2]; by = cwall->vertices[3];
            }else{
                bx = cwall->vertices[0]; by = cwall->vertices[1];
                ax = cwall->vertices[2]; ay = cwall->vertices[3];
            }
            if(ax < 0){
                if(ay<by){
                    float k=-cwall->b/cwall->a; float b=-cwall->c/cwall->a;
                    for(int y=ay; y<by; y++){
                        for(int x=ax; x<k*y+b; x++){

                        }
                    }
                }

            }

        }
    }*/
}
__global__
void Do_Metropolis_Hastings(sharedWrapper *gWrapper, float * gArray){
    sWrapper[0] = *gWrapper;
    if(blockIdx.x == 0 ){
        Initialize_Room_In_Device(sWrapper[0].wRoom, sWrapper[0].wObjs, sWrapper[0].initialMask,sWrapper[0].obstacleVertices,sWrapper[0].wFloats);
    }
    else{
        if(threadIdx.x < sWrapper[0].wRoom->objctNum){
            int objId = blockIdx.x * sWrapper[0].wRoom->objctNum + threadIdx.x;
            sWrapper[0].wObjs[objId] = sWrapper[0].wObjs[threadIdx.x];
        }
        int baseId = blockIdx.x *sWrapper[0].wRoom->mskCount;
        for(int i=threadIdx.x; i<sWrapper[0].wRoom->rowCount; i+=nThreads){
            for(int j=0;j<sWrapper[0].wRoom->colCount;j++)
                sWrapper[0].wMask[baseId + i * sWrapper[0].wRoom->colCount + j] = sWrapper[0].wMask[i * sWrapper[0].wRoom->colCount + j];
        }
    }

    float* costList = sWrapper[0].wFloats;
    float* temparature = (float *) & costList[nBlocks * nThreads];
    int * pickedupIds = (int*) &temparature[nBlocks];

    temparature[blockIdx.x] = get_float_random(10)/100;
    for(int i=threadIdx.x; i<gWrapper->nTimes; i+=nThreads)
        pickedupIds[i] = sWrapper[0].wRoom->freeObjIds[get_int_random(sWrapper[0].wRoom->freeObjNum)];


    Metropolis_Hastings(costList, temparature, pickedupIds);
    // if(blockIdx.x == 0)
    // printf("thread: %d, err: \n",threadIdx.x, hipGetLastError());
    __syncthreads();

    if(threadIdx.x < sWrapper[0].wRoom->objctNum){
        int singleSize = gWrapper->wRoom->objctNum * 4 + 1;
        for(int i=0; i<MAX_KEPT_RES; i++){
            gWrapper->resTransAndRot[singleSize * i] =  sWrapper[0].resTransAndRot[singleSize * i];

            int startPos = singleSize*i + 4*threadIdx.x;
            for(int k=1; k<5; k++)
                gWrapper->resTransAndRot[startPos + k] = sWrapper[0].resTransAndRot[startPos + k];
        }
    }
    __syncthreads();


    //gArray[threadIdx.x] = costList[threadIdx.x];
}

void generate_suggestions(Room * m_room, int nTimes){
    sharedWrapper *gWrapper;
    hipMallocManaged(&gWrapper,  sizeof(sharedWrapper));

    gWrapper->nTimes = nTimes;
    hipMallocManaged(&gWrapper->wRoom, sizeof(sharedRoom));
    m_room->CopyToSharedRoom(gWrapper->wRoom);

    int objMem = nBlocks *m_room->objctNum * sizeof(singleObj);
    hipMallocManaged(&gWrapper->wObjs, objMem);
	for(int i=0; i<m_room->objctNum; i++){
        gWrapper->wObjs[i] = m_room->objects[i];
        hipMallocManaged(&gWrapper->wObjs[i].objMask, m_room->objects[i].maskLen* m_room->objects[i].maskLen* sizeof(unsigned char));
        hipMemset(gWrapper->wObjs[i].objMask, 0,  m_room->objects[i].maskLen* m_room->objects[i].maskLen);
    }

    // Initial three masks
    int tMem = gWrapper->wRoom->mskCount * sizeof(unsigned char);

    hipMallocManaged(&gWrapper->initialMask, tMem);
    hipMemset(gWrapper->initialMask, 0, tMem);

    hipMallocManaged(&gWrapper->wMask, nBlocks *tMem);

    hipMallocManaged(&gWrapper->backMask, nBlocks *tMem);
    hipMemset(gWrapper->backMask, 0, nBlocks*tMem);

    if(m_room->obstacles.size()!=0){
        int obstacleVerticesMem = sizeof(float) * 8 * m_room->obstacles.size();
        hipMallocManaged(&gWrapper->obstacleVertices, obstacleVerticesMem);
        m_room->get_obstacle_vertices(gWrapper->obstacleVertices);
    }

	int floatMem =  (nBlocks *(2+nThreads)) * sizeof(float);
    hipMallocManaged(&gWrapper->wFloats, floatMem);
    int maskAreaMem = 2*nBlocks * sizeof(float);
    hipMallocManaged(&gWrapper->wmaskArea, maskAreaMem);
    hipMemset(gWrapper->wmaskArea, 0, maskAreaMem);

    int pairMem = m_room->actualPairs.size() * 4 * sizeof(int);
    hipMallocManaged(&gWrapper->wPairRelation, pairMem);
    for(int i=0;i<m_room->actualPairs.size();i++){
        for(int j=0;j<4;j++)
            gWrapper->wPairRelation[4*i+j]= m_room->actualPairs[i][j];
    }

    int resMem = (m_room->objctNum * 4 + 1) * MAX_KEPT_RES * sizeof(float);
    hipMallocManaged(&gWrapper->resTransAndRot, resMem);

    float * gArray;
    hipMallocManaged(&gArray, nThreads * sizeof(float));
    const char * errStr1 = hipGetErrorString (hipGetLastError());
    cout<<"error:"<<errStr1<<endl;
	Do_Metropolis_Hastings<<<nBlocks, nThreads, sizeof(*gWrapper)>>>(gWrapper, gArray);
	hipDeviceSynchronize();
    const char * errStr = hipGetErrorString (hipGetLastError());
    cout<<"error:"<<errStr<<endl;
    int singleSize = 4*m_room->objctNum + 1;
    for(int i=0, startId = 0; i< MAX_KEPT_RES; i++, startId = i*singleSize){
        cout<<"result: "<<i<<"- cost: "<<gWrapper->resTransAndRot[startId]<<endl;

        for(int n=0; n<m_room->objctNum; n++){
            cout<<"object: "<<n<<" pos and rot:";
            string res = "";
            for(int pi=1;pi<5;pi++)
                res+= to_string(gWrapper->resTransAndRot[startId+4*n+pi]) + " ";
            cout<<res<<endl;
        }
    }
    display_suggestions(m_room, gWrapper->resTransAndRot);
    hipFree(gWrapper->wRoom);
    hipFree(gWrapper->wObjs);
    hipFree(gWrapper->initialMask);
    hipFree(gWrapper->wMask);
    hipFree(gWrapper->backMask);
    hipFree(gWrapper->obstacleVertices);
    hipFree(gWrapper->wFloats);
    hipFree(gWrapper->wPairRelation);
    hipFree(gWrapper->resTransAndRot);
}

void startToProcess(Room * m_room, int nTimes){
    if(m_room->objctNum == 0)
        return;
	setUpDevices();

    clock_t start, finish;
    float costtime;
    start = clock();
	generate_suggestions(m_room, nTimes);

    finish = clock();
    costtime = (float)(finish - start) / CLOCKS_PER_SEC;
    cout<<"Runtime: "<<costtime<<endl;
}

int main(int argc, char** argv){
    //char* filename;
    int nTimes = DEFAULT_RUN_TIMES;
    for(int i=1; i<argc; i++){
        if(argv[i][0] == '-'){
            switch (argv[i][1]) {
                case 'n':
                    nTimes = (int)strtol(argv[i+1], (char **)nullptr, 10);
                    break;
            }
        }
    }
	//char* existance_file;
	//filename = new char[100];
	//existance_file = new char[100];
	//int r = strcpy_s(filename, 100, "E:/layoutParam.txt");
	//r = strcpy_s(existance_file, 100, "E:/fixedObj.txt");
	Room* parserRoom = new Room();
    setupDebugRoom(parserRoom);

	// parser_inputfile(filename, parserRoom);
	// parser_inputfile(existance_file, room, weights);
	// if (parserRoom != nullptr && (parserRoom->objctNum != 0 || parserRoom->wallNum != 0))
    startToProcess(parserRoom, nTimes);
	return 0;
}
