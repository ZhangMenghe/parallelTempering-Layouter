#include "hip/hip_runtime.h"
#include<iostream>
#include "room.cuh"

#include "math.h"

using namespace std;
__device__ __host__
void rot_around_point(float center[3], float * x, float * y, float s, float c) {
	// translate point back to origin:
	*x -= center[0];
	*y -= center[1];

	// rotate point
	float xnew = *x * c - *y * s;
	float ynew = *x * s + *y * c;

	// translate point back:
	*x = xnew + center[0];
	*y = ynew + center[1];
}
//ax,ay,bx,by
void Room::init_a_wall(wall *newWall, vector<float> params) {
	float ax = params[0], ay = params[1], bx = params[2], by = params[3];
	newWall->translation[0] = (ax + bx) / 2;newWall->translation[1] = (ay + by) / 2;newWall->translation[2] = 0;
	newWall->width = sqrtf(powf((by - ay), 2) + powf((bx - ax), 2));
	copy(params.begin(), params.end(), newWall->vertices);
	if (ax == bx) {
		newWall->zrotation =PI/2;
		newWall->b = 0; newWall->a = 1; newWall->c = -ax;
	}
	else if (ay == by) {
		newWall->zrotation = 0;
		newWall->a = 0; newWall->b = 1; newWall->c = -ay;
	}
	else {
		newWall->a = (by - ay) / (bx - ax); newWall->b = -1; newWall->c = -(newWall->a*ax - ay);
		newWall->zrotation = atanf(newWall->a)/ PI;
	}
}
// 4*2 vertices, 2 center, 2 size, angle, label, zheight
void Room::init_an_object(vector<float>params, bool isFixed, bool isPrevious) {
	singleObj obj;
	obj.id = objects.size();
	//vertices
	copy(params.begin(), params.begin() + 8, obj.vertices);

	obj.translation[0] = params[8];obj.translation[1] =params[9];obj.translation[2] =.0f;
	obj.objWidth = params[10];
	obj.objHeight = params[11];
	set_obj_zrotation(&obj, params[12] * ANGLE_TO_RAD_F);
	//obj.zrotation = params[12] * ANGLE_TO_RAD_F;
	obj.catalogId = params[13];
	obj.zheight = params[14];
	obj.area = obj.objWidth * obj.objHeight;
	obj.isFixed = isFixed;
	obj.alignedTheWall = (obj.catalogId == TYPE_SHELF || obj.catalogId == TYPE_BED || obj.catalogId == TYPE_TABLE) ? true : false;
	obj.adjoinWall = (obj.catalogId == TYPE_SHELF || obj.catalogId == TYPE_BED || obj.catalogId == TYPE_TABLE) ? true : false;

	// TODO: is it necessary?
	// if (!isPrevious)//existing objs' values should be
		// update_obj_boundingBox_and_vertices(obj, 0);

	//move this calculation to device
	indepenFurArea += obj.objWidth * obj.objHeight; //get_single_obj_maskArea(obj.vertices);

	obj.maskLen = int(sqrtf(obj.objWidth * obj.objWidth + obj.objHeight * obj.objHeight));

	int gidx = 0;
	for(; gidx<groupNum; gidx++){
		if(groupMap[gidx].gid == params[15]){
			groupMap[gidx].objIds[groupMap[gidx].memNum++] = obj.id;
			break;
		}

	}
	if(gidx == groupNum){
		groupMap[groupNum].gid = params[15];
		groupMap[groupNum].memNum = 1;
		groupMap[groupNum].objIds[0] = obj.id;
		groupNum++;
	}

	objects.push_back(obj);
	objctNum++;
	if (!isFixed)
		freeObjIds[freeObjNum++] = obj.id;
}
void Room::set_pairwise_map() {
	pairMap[0].pid = TYPE_CHAIR;
	int mtype[3] = {TYPE_CHAIR, TYPE_COFFETABLE, TYPE_ENDTABLE};
	copy(begin(mtype), end(mtype), begin(pairMap[0].objTypes));
	int mdist[3] = {0,40,0}; int mdistm[3] =  {50,46,30};
	copy(begin(mdist), end(mdist), begin(pairMap[0].minDist));
	copy(begin(mdistm), end(mdistm), begin(pairMap[0].maxDist));

	pairMap[1].pid = TYPE_BED;
	pairMap[1].objTypes[0] = TYPE_NIGHTSTAND;
	pairMap[1].minDist[0] = 0;
	pairMap[1].maxDist[0] = 30;
}
void Room::set_objs_pairwise_relation(const singleObj& obj1, const singleObj& obj2){
	const singleObj* indexObj = (obj1.catalogId <= obj2.catalogId)?&obj1:&obj2;
	const singleObj* compObj = (obj1.id == indexObj->id)? &obj2:&obj1;
	for(int i=0; i<CONSTRAIN_PAIRS; i++){
		if(indexObj->catalogId == pairMap[i].pid){
			for(int j=0; pairMap[i].objTypes[j]!=-1&&j<MAX_SUPPORT_TYPE; j++){
				if(pairMap[i].objTypes[j] == compObj->catalogId){
					vector<int> pair{indexObj->id, compObj->id, pairMap[i].minDist[j],  pairMap[i].maxDist[j]};
					actualPairs.push_back(pair);
					break;
				}
			}
		}
	}
}
void Room::update_mask_by_wall(const wall* wal) {
	//TODO: DON'T KNOW HOW TO TACKLE WITH OBLIQUE WALL
}



void Room::CopyToSharedRoom(sharedRoom *m_room){
	m_room->objctNum = objctNum;
	m_room->wallNum = wallNum;
	m_room->obstacleNum = obstacles.size();
	m_room->freeObjNum = freeObjNum;
	m_room->half_width = half_width;
	m_room->half_height = half_height;
	m_room->indepenFurArea = indepenFurArea;
	m_room->obstacleArea = obstacleArea;
	m_room->wallArea = wallArea;
	m_room->overlappingThreshold = overlappingThreshold;
	m_room->colCount = colCount;
	m_room->rowCount = rowCount;
	m_room->mskCount = colCount * rowCount;
	m_room->pairNum = actualPairs.size();
	m_room->groupNum = groupNum;
	m_room->RoomCenter[0] = center[0];m_room->RoomCenter[1] = center[1];m_room->RoomCenter[2] = center[2];

	hipMemcpy(m_room->freeObjIds, freeObjIds, freeObjNum* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(m_room->groupMap, groupMap, MAX_GROUP_ALLOW* sizeof(groupMapStruct), hipMemcpyHostToDevice);
	hipMemcpy(m_room->pairMap, pairMap, CONSTRAIN_PAIRS* sizeof(pairMapStruct), hipMemcpyHostToDevice);
	for(int i=0;i<wallNum;i++)
		m_room->deviceWalls[i] = walls[i];
	//
	//
	// int tMem = colCount*rowCount * sizeof(unsigned char);
	// hipMallocManaged(&furnitureMask, tMem);
	// hipMallocManaged(&furnitureMask_initial, tMem);
	// hipMemcpy(furnitureMask, m_room->furnitureMask, tMem, hipMemcpyHostToDevice);
	// hipMemcpy(furnitureMask_initial, m_room->furnitureMask_initial, tMem, hipMemcpyHostToDevice);
	// //TODO:obstacle
}

void Room::initialize_room(float s_width, float s_height) {
	initialized = true;
	groupNum = 0;
	half_width = s_width / 2;
	half_height = s_height / 2;
	overlappingThreshold = s_width * s_height * 0.005;
	set_pairwise_map();
	rowCount = int(s_height) + 1;	colCount = int(s_width)+1;
	int tMem = rowCount * colCount * sizeof(unsigned char);
	furnitureMask_initial = (unsigned char *)malloc(tMem);
	memset(furnitureMask_initial, (unsigned char)0, colCount*rowCount);
}
void Room::add_a_wall(vector<float> params){
	wall newWall;
	newWall.id = walls.size();
	newWall.zheight = params[4];
	init_a_wall(&newWall, params);
	walls.push_back(newWall);
	wallNum++;
	if (fabs(fmod(newWall.zrotation, PI)) > 0.01)
		update_mask_by_wall(&newWall);
}
void Room::add_an_object(vector<float> params, bool isPrevious, bool isFixed) {
	if (params.size() < 15) {
		float hw = params[2] / 2, hh = params[3] / 2;
		float cx = params[0], cy = params[1];
		float res[8] = { -hw + cx, hh + cy, hw + cx, hh + cy, hw + cx, -hh + cy, -hw + cx, -hh + cy };
		vector<float>vertices(res, res + 8);// get_vertices_by_pos(params[0], params[1], params[2] / 2, params[3] / 2);
		params.insert(params.begin(), vertices.begin(), vertices.end());
	}
	if (isPrevious) {
		switch (int(params[13]))
		{
		case 1:
			params[13] = TYPE_FLOOR;
			break;
		case 3://chair
			params[13] = TYPE_CHAIR;
			break;
		case 8:
			params[13] = TYPE_WALL;
			break;
		case 10:
			params[13] = TYPE_OTHER;
			break;
		case 11:
			params[13] = TYPE_CEILING;
			break;
		}
	}
	//default groupid is 0
	if(params.size()<16)
		params.push_back(0);

	init_an_object(params, isFixed, isPrevious);
}
void Room::add_a_focal_point(vector<float> fp) {
	int groupId = (fp.size() == 3)? 0:fp[3];
	for(int i=0; i<groupNum; i++){
		if(groupId == groupMap[i].gid)
			copy(fp.begin(), fp.begin()+3, groupMap[i].focal);
	}
}


void Room::set_obj_zrotation(singleObj * obj, float nrot) {
	float oldRot = obj->zrotation;
	nrot = remainderf(nrot, 2*PI);
	obj->zrotation = nrot;
	float gap = obj->zrotation - oldRot;
	float s = sinf(gap); float c=cosf(gap);
	float minx = INFINITY,maxx =-INFINITY, miny=INFINITY, maxy = -INFINITY;
	for(int i=0; i<4; i++){
		rot_around_point(obj->translation, &obj->vertices[2*i], &obj->vertices[2*i+1], s, c);
		minx = (obj->vertices[2*i] < minx)? obj->vertices[2*i]:minx;
		maxx = (obj->vertices[2*i] > maxx)? obj->vertices[2*i]:maxx;
		miny = (obj->vertices[2*i + 1] < miny)? obj->vertices[2*i+1]:miny;
		maxy = (obj->vertices[2*i + 1] > maxy)? obj->vertices[2*i+1]:maxy;
	}
	obj->boundingBox.x = minx; obj->boundingBox.y=maxy;
	obj->boundingBox.width = maxx-minx; obj->boundingBox.height = maxy-miny;
}


	void Room::add_an_obstacle(vector<float> params) {
		obstacles.push_back(params);
	}
	void Room::get_obstacle_vertices(float * vertices){
		for(int i=0; i<obstacles.size();i++)
			copy(obstacles[i].begin(), obstacles[i].end(),&vertices[8*i]);
	}
// 	void change_obj_freeState(singleObj* obj) {
// 		if (obj->isFixed)
// 			freeObjIds.erase(remove(freeObjIds.begin(), freeObjIds.end(), obj->id));
// 		else
// 			freeObjIds.push_back(obj->id);
// 		obj->isFixed = !obj->isFixed;
// 	}
// };
